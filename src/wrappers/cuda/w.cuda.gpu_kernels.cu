
#include <hip/hip_runtime.h>

#ifdef HAVE_CUDA

#include "gpu/gpu_kernels.h"

#include <complex>

#include "w.cuda.common.h"



namespace espreso {
namespace gpu {
namespace kernels {

    namespace
    {
        template<typename T>
        static __device__ void complexAtomicAdd(std::complex<T> * dst, std::complex<T> val)
        {
            atomicAdd(&reinterpret_cast<T*>(dst)[0], reinterpret_cast<T*>(&val)[0]);
            atomicAdd(&reinterpret_cast<T*>(dst)[1], reinterpret_cast<T*>(&val)[1]);
        }

        template<typename T>
        static __device__ void myAtomicAdd(T * dst, T val) { atomicAdd(dst, val); }
        template<typename T>
        static __device__ void myAtomicAdd(std::complex<T> * dst, std::complex<T> val) { complexAtomicAdd(dst, val); }

        template<typename T, typename I>
        static __global__ void _do_DCmap_scatter(T ** domain_vectors, const I * n_dofs_interfaces, const T * cluster_vector, I const * const * D2Cs)
        {
            // one block per domain
        
            I d = blockIdx.x;
            I n_dofs_interface = n_dofs_interfaces[d];
            T * domain_vector = domain_vectors[d];
            const I * D2C = D2Cs[d];
        
            for(I dof = threadIdx.x; dof < n_dofs_interface; dof += blockDim.x)
            {
                domain_vector[dof] = cluster_vector[D2C[dof]];
            }
        }
        
        template<typename T, typename I>
        static __global__ void _do_DCmap_gather(T const * const * domain_vectors, const I * n_dofs_interfaces, T * cluster_vector, I const * const * D2Cs)
        {
            // one block per domain
        
            I d = blockIdx.x;
            I n_dofs_interface = n_dofs_interfaces[d];
            const T * domain_vector = domain_vectors[d];
            const I * D2C = D2Cs[d];
        
            for(I dof = threadIdx.x; dof < n_dofs_interface; dof += blockDim.x)
            {
                myAtomicAdd(&cluster_vector[D2C[dof]], domain_vector[dof]);
            }
        }
    }

    template<typename T, typename I>
    void DCmap_scatter(mgm::queue & q, Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, const Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs)
    {
        I n_domains = domain_vector_pointers.size;
        _do_DCmap_scatter<T,I><<< n_domains, 256, 0, q.stream >>>(domain_vector_pointers.vals, n_dofs_interfaces.vals, cluster_vector.vals, D2Cs.vals);
        CHECK(cudaPeekAtLastError());
    }

    template<typename T, typename I>
    void DCmap_gather(mgm::queue & q, const Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs)
    {
        I n_domains = domain_vector_pointers.size;
        _do_DCmap_gather<T,I><<< n_domains, 256, 0, q.stream >>>(domain_vector_pointers.vals, n_dofs_interfaces.vals, cluster_vector.vals, D2Cs.vals);
        CHECK(cudaPeekAtLastError());
    }



    #define INSTANTIATE(T,I) \
    template void DCmap_scatter<T,I>(mgm::queue & q, Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, const Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs); \
    template void DCmap_gather<T,I>(mgm::queue & q, const Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs);

    INSTANTIATE(float,  int32_t)
    INSTANTIATE(double, int32_t)
    INSTANTIATE(float,  int64_t)
    INSTANTIATE(double, int64_t)
    INSTANTIATE(std::complex<float>,  int32_t)
    INSTANTIATE(std::complex<double>, int32_t)
    INSTANTIATE(std::complex<float>,  int64_t)
    INSTANTIATE(std::complex<double>, int64_t)

    #undef INSTANTIATE
}
}
}

#endif
