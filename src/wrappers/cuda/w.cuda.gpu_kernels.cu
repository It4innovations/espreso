
#include <hip/hip_runtime.h>

#ifdef HAVE_CUDA

#include "gpu/gpu_kernels.h"

#include "w.cuda.common.h"



namespace espreso {
namespace gpu {
namespace kernels {

    namespace
    {
        template<typename T, typename I>
        static __global__ void _do_DCmap_scatter(T ** domain_vectors, const I * n_dofs_interfaces, const T * cluster_vector, I const * const * D2Cs)
        {
            // one block per domain
        
            I d = blockIdx.x;
            I n_dofs_interface = n_dofs_interfaces[d];
            T * domain_vector = domain_vectors[d];
            const I * D2C = D2Cs[d];
        
            for(I dof = threadIdx.x; dof < n_dofs_interface; dof += blockDim.x)
            {
                domain_vector[dof] = cluster_vector[D2C[dof]];
            }
        }
        
        template<typename T, typename I>
        static __global__ void _do_DCmap_gather(T const * const * domain_vectors, const I * n_dofs_interfaces, T * cluster_vector, I const * const * D2Cs)
        {
            // one block per domain
        
            I d = blockIdx.x;
            I n_dofs_interface = n_dofs_interfaces[d];
            const T * domain_vector = domain_vectors[d];
            const I * D2C = D2Cs[d];
        
            for(I dof = threadIdx.x; dof < n_dofs_interface; dof += blockDim.x)
            {
                atomicAdd(&cluster_vector[D2C[dof]], domain_vector[dof]);
            }
        }
    }

    template<typename T, typename I>
    void DCmap_scatter(mgm::queue & q, Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, const Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs)
    {
        I n_domains = domain_vector_pointers.size;
        _do_DCmap_scatter<T,I><<< n_domains, 256, 0, q.stream >>>(domain_vector_pointers.vals, n_dofs_interfaces.vals, cluster_vector.vals, D2Cs.vals);
        CHECK(cudaPeekAtLastError());
    }

    template<typename T, typename I>
    void DCmap_gather(mgm::queue & q, const Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs)
    {
        I n_domains = domain_vector_pointers.size;
        _do_DCmap_gather<T,I><<< n_domains, 256, 0, q.stream >>>(domain_vector_pointers.vals, n_dofs_interfaces.vals, cluster_vector.vals, D2Cs.vals);
        CHECK(cudaPeekAtLastError());
    }



    #define INSTANTIATE(T,I) \
    template void DCmap_scatter<T,I>(mgm::queue & q, Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, const Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs); \
    template void DCmap_gather<T,I>(mgm::queue & q, const Vector_Dense<T*,I,mgm::Ad> & domain_vector_pointers, const Vector_Dense<I,I,mgm::Ad> & n_dofs_interfaces, Vector_Dense<T,I,mgm::Ad> & cluster_vector, const Vector_Dense<I*,I,mgm::Ad> & D2Cs);

    INSTANTIATE(float,  int32_t)
    INSTANTIATE(float,  int64_t)
    INSTANTIATE(double, int32_t)
    INSTANTIATE(double, int64_t)

    #undef INSTANTIATE
}
}
}




#endif
