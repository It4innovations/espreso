
#include <hip/hip_runtime.h>

#ifdef HAVE_CUDA

#include "w.cuda.gpu_spblas_common.h"
#include "w.cuda.gpu_management.h"

#include <type_traits>
#include <algorithm>
#include <complex>
#include <cub/cub.cuh>



namespace espreso {
namespace gpu {
namespace spblas {

    namespace {
        template<typename I>
        I get_most_significant_bit(I val)
        {
            static_assert(std::is_integral_v<I> && std::is_unsigned_v<I>, "wrong type");
            I msb = 0;
            while(val != 0)
            {
                val >>= 1;
                msb++;
            }
            return msb;
        }

        template<typename T> __device__ constexpr bool is_complex();
        template<> __device__ constexpr bool is_complex<int32_t>() { return false; }
        // template<> __device__ constexpr bool is_complex<int64_t>() { return false; }
        // template<> __device__ constexpr bool is_complex<float>() { return false; }
        template<> __device__ constexpr bool is_complex<double>() { return false; }
        // template<> __device__ constexpr bool is_complex<std::complex<float>>() { return true; }
        // template<> __device__ constexpr bool is_complex<std::complex<double>>() { return true; }

        template<typename T>
        static __global__ void _init_linear(T * output, size_t count)
        {
            size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
            size_t stride = blockDim.x * gridDim.x;
            for(size_t i = idx; i < count; i += stride) output[i] = (T)i;
        }

        template<typename T, typename I, bool conj = false>
        static __global__ void _permute_array(T * output, T const * input, I const * perm, size_t count)
        {
            size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
            size_t stride = blockDim.x * gridDim.x;
            for(size_t i = idx; i < count; i += stride)
            {
                T & out = output[i];
                const T & val = input[perm[i]];
                if constexpr(conj && is_complex<T>())
                {
                    reinterpret_cast<T*>(&out)[0] =  reinterpret_cast<T*>(&val)[0];
                    reinterpret_cast<T*>(&out)[1] = -reinterpret_cast<T*>(&val)[1];
                }
                else
                {
                    out = val;
                }
            }
        }

        template<typename I>
        static __global__ void _csr_to_ijv_rowidxs(I * out_rowidxs, I const * in_rowptrs)
        {
            I r = blockIdx.x;
            I start = in_rowptrs[r];
            I end = in_rowptrs[r+1];
            for(I i = start + threadIdx.x; i < end; i += blockDim.x) out_rowidxs[i] = r;
        }
    }

    template<typename I>
    void my_csr_transpose_buffersize(cudaStream_t & stream, I input_nrows, I input_ncols, I nnz, size_t & buffersize)
    {
        I output_nrows = input_ncols;
        I end_bit = get_most_significant_bit((uint64_t)input_ncols);
        size_t bfs_map, bfs_linear, bfs_sorted_colidxs;
        size_t bfs_hist, bfs_scan, bfs_sort;
        bfs_map = nnz * sizeof(I);
        bfs_linear = nnz * sizeof(I);
        bfs_sorted_colidxs = nnz * sizeof(I);
        CHECK(cub::DeviceHistogram::HistogramEven(nullptr, bfs_hist, (I*)nullptr, (I*)nullptr, output_nrows+1, 0, output_nrows, nnz, stream));
        CHECK(cub::DeviceScan::ExclusiveSum(nullptr, bfs_scan, (I*)nullptr, (I*)nullptr, output_nrows+1, stream));
        CHECK(cub::DeviceRadixSort::SortPairs(nullptr, bfs_sort, (I*)nullptr, (I*)nullptr, (I*)nullptr, (I*)nullptr, nnz, 0, end_bit, stream));
        CHECK(cudaStreamSynchronize(stream));
        buffersize = bfs_map + bfs_linear + bfs_sorted_colidxs + std::max(std::max(bfs_hist, bfs_scan), bfs_sort);
    }
    
    template<typename I>
    void my_csr_transpose_preprocess(cudaStream_t & stream, I input_nrows, I input_ncols, I nnz, I * input_rowptrs, I * input_colidxs, I * output_rowptrs, I * output_colidxs, size_t buffersize, void * buffer)
    {
        I output_nrows = input_ncols;
        I end_bit = get_most_significant_bit((uint64_t)input_ncols);
        I * map = (I*)buffer;
        buffer = (char*)buffer + nnz * sizeof(I);   buffersize -= nnz * sizeof(I);
        I * linear = (I*)buffer;
        buffer = (char*)buffer + nnz * sizeof(I);   buffersize -= nnz * sizeof(I);
        I * colidxs_sorted = (I*)buffer;
        buffer = (char*)buffer + nnz * sizeof(I);   buffersize -= nnz * sizeof(I);
        CHECK(cub::DeviceHistogram::HistogramEven(buffer, buffersize, input_colidxs, output_rowptrs, output_nrows+1, 0, output_nrows, nnz, stream));
        CHECK(cub::DeviceScan::ExclusiveSum(buffer, buffersize, output_rowptrs, output_rowptrs, output_nrows+1, stream));
        _init_linear<<< 16, 256, 0, stream >>>(linear, nnz);
        CHECK(cudaPeekAtLastError());
        CHECK(cub::DeviceRadixSort::SortPairs(buffer, buffersize, input_colidxs, colidxs_sorted, linear, map, nnz, 0, end_bit, stream));
        I * ijv_rowidxs = colidxs_sorted; // just two unrelated temporary buffers sharing the same memory
        _csr_to_ijv_rowidxs<<< input_nrows, 64, 0, stream >>>(ijv_rowidxs, input_rowptrs);
        CHECK(cudaPeekAtLastError());
        _permute_array<<< 16, 256, 0, stream >>>(output_colidxs, ijv_rowidxs, map, nnz);
        CHECK(cudaPeekAtLastError());
    }
    
    template<typename T, typename I>
    void my_csr_transpose_compute(cudaStream_t & stream, I nnz, T * input_vals, T * output_vals, bool conjugate, void * buffer)
    {
        I * map = (I*)buffer;
        if(conjugate) _permute_array<T,I,true> <<< 16, 256, 0, stream >>>(output_vals, input_vals, map, nnz);
        else          _permute_array<T,I,false><<< 16, 256, 0, stream >>>(output_vals, input_vals, map, nnz);
        CHECK(cudaPeekAtLastError());
    }



    #define INSTANTIATE_T_I(T,I) \
    template void my_csr_transpose_compute<T,I>(cudaStream_t & stream, I nnz, T * input_vals, T * output_vals, bool conjugate, void * buffer);

        #define INSTANTIATE_I(I) \
        /* INSTANTIATE_T_I(float,I) */ \
        INSTANTIATE_T_I(double,I) \
        /* INSTANTIATE_T_I(std::complex<float>,I) */ \
        /* INSTANTIATE_T_I(std::complex<double>,I) */ \
        template void my_csr_transpose_buffersize<I>(cudaStream_t & stream, I input_nrows, I input_ncols, I nnz, size_t & buffersize); \
        template void my_csr_transpose_preprocess<I>(cudaStream_t & stream, I input_nrows, I input_ncols, I nnz, I * input_rowptrs, I * input_colidxs, I * output_rowptrs, I * output_colidxs, size_t buffersize, void * buffer);

            INSTANTIATE_I(int32_t)
            // INSTANTIATE_I(int64_t)

        #undef INSTANTIATE_I
    #undef INSTANTIATE_T_I

}
}
}

#endif
